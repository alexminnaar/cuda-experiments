
/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <iostream>
#include <ctime>
#include <hip/hip_runtime.h>
				// Stops underlining of __global__
	// Stops underlining of threadIdx etc.

using namespace std;


__global__ void FindClosestGPU(float3* points, int* indices, int count)
{
if(count <= 1) return;

int idx = threadIdx.x + blockIdx.x * blockDim.x;
if(idx < count)
	{
	float3 thisPoint = points[idx];
	float smallestSoFar = 3.40282e38f;

	for(int i = 0; i < count; i++)
		{
		if(i == idx) continue;

		float dist = (thisPoint.x - points[i].x)*(thisPoint.x - points[i].x);
		dist += (thisPoint.y - points[i].y)*(thisPoint.y - points[i].y);
		dist += (thisPoint.z - points[i].z)*(thisPoint.z - points[i].z);

		if(dist < smallestSoFar)
			{
			smallestSoFar = dist;
			indices[idx] = i;
			}
		}
	}
}



int main(int argc, char **argv)
{
	cout<<"running GPU implementation"<<endl;
	// Number of points
	const int count = 10000;

	// Arrays of points
	int *indexOfClosest = new int[count];
	float3 *points = new float3[count];
	float3* d_points;	 // GPU version
	int* d_indexOfClosest;

	// Create a list of random points
	for(int i = 0; i < count; i++)
		{
		points[i].x = (float)((rand()%10000) - 5000);
		points[i].y = (float)((rand()%10000) - 5000);
		points[i].z = (float)((rand()%10000) - 5000);
		}

	hipMalloc(&d_points, sizeof(float3) * count);
	hipMemcpy(d_points, points, sizeof(float3) * count, hipMemcpyHostToDevice);
	hipMalloc(&d_indexOfClosest, sizeof(int) * count);

	// This variable is used to keep track of the fastest time so far
	long fastest = 1000000;

	// Run the algorithm 20 times
	for(int q = 0; q < 20; q++)
		{
		long startTime = clock();

		// Run the algorithm
		//FindClosestCPU(points, indexOfClosest, count);

		FindClosestGPU<<<(count / 320)+1, 320>>>(d_points, d_indexOfClosest, count);
		hipMemcpy(indexOfClosest, d_indexOfClosest, sizeof(int) * count, hipMemcpyDeviceToHost);

		long finishTime = clock();

		cout<<q<<" "<<(finishTime - startTime)<<endl;

		// If that run was faster update the fastest time so far
		if((finishTime - startTime) < fastest)
			fastest = (finishTime - startTime);
		}

	// Print out the fastest time
	cout<<"Fastest time: "<<fastest<<endl;

	// Print the final results to screen
	cout<<"Final results:"<<endl;
	for(int i = 0; i < 10; i++)
		cout<<i<<"."<<indexOfClosest[i]<<endl;

	// Deallocate ram
	delete[] indexOfClosest;
	delete[] points;
	hipFree(d_points);
	hipFree(d_indexOfClosest);

	hipDeviceReset();

	return 0;
}
